#include "hip/hip_runtime.h"
#include "../include/quotients.cuh"

#include <cstdio>

#define THREAD_COUNT_MAX 1024 

typedef struct {
    secure_field_point point;
    uint32_t *columns;
    qm31 *values;
    uint32_t size;
} column_sample_batch;

__device__ point index_to_point(uint32_t index) {
    return point_pow(m31_circle_gen, (int)index);
}

__device__ point domain_at_index(uint32_t half_coset_initial_index, uint32_t half_coset_step_size, uint32_t index, uint32_t domain_size) {
    uint32_t half_coset_size = domain_size >> 1;

    if (index < half_coset_size) {
        int modulo_u31_mask = 0x7fffffff;
        uint64_t global_index = (uint64_t) half_coset_initial_index + (uint64_t) half_coset_step_size * (uint64_t) index;
        return index_to_point(global_index & modulo_u31_mask);
    } else {
        int modulo_u31_mask = 0x7fffffff;
        uint64_t global_index = (uint64_t) half_coset_initial_index + (uint64_t) half_coset_step_size * (uint64_t) (index - half_coset_size);
        return index_to_point((2147483648 - global_index) & modulo_u31_mask);
    }
}

void column_sample_batches_for(
        secure_field_point *sample_points,
        uint32_t *sample_column_indexes,
        qm31 *sample_column_values,
        const uint32_t *sample_column_and_values_sizes,
        uint32_t sample_size,
        column_sample_batch *result
) {
    unsigned int offset = 0;
    for (unsigned int index = 0; index < sample_size; index++) {
        result[index].point = sample_points[index];
        result[index].columns = &sample_column_indexes[offset];
        result[index].values = &sample_column_values[offset];
        result[index].size = sample_column_and_values_sizes[index];
        offset += sample_column_and_values_sizes[index];
    }
};

__device__ void complex_conjugate_line_coeffs(secure_field_point point, qm31 value, qm31 alpha, qm31* a_out, qm31* b_out, qm31* c_out) {
    qm31 a = sub(qm31{value.a, neg(value.b)}, value); 
    qm31 c = sub(qm31{point.y.a, neg(point.y.b)}, point.y);
    qm31 b = sub(mul(value, c), mul(a, point.y));  

    *a_out = mul(alpha, a);
    *b_out = mul(alpha, b);
    *c_out = mul(alpha, c);
}

__global__ void column_line_and_batch_random_coeffs(
    column_sample_batch *sample_batches,
    uint32_t sample_size,
    qm31 random_coefficient,
    qm31 *flattened_line_coeffs,
    uint32_t *line_coeffs_sizes,
    qm31 *batch_random_coeffs
) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if(tid < sample_size) {
        // Calculate Batch Random Coeffs
        batch_random_coeffs[tid] = pow(random_coefficient, sample_batches[tid].size); 

        // Calculate Column Line Coeffs
        line_coeffs_sizes[tid] = sample_batches[tid].size;
        size_t sample_batches_offset = tid * line_coeffs_sizes[tid] * 3; 

        qm31 alpha = qm31{cm31{m31{1}, m31{0}}, cm31{m31{0}, m31{0}}};

        for(size_t j = 0; j < sample_batches[tid].size; ++j) {
            qm31 sampled_value = sample_batches[tid].values[j];
            alpha = mul(alpha, random_coefficient); 
            secure_field_point point = sample_batches[tid].point;
            qm31 value = sampled_value; 

            size_t sampled_offset = sample_batches_offset + (j * 3);
            complex_conjugate_line_coeffs(point, value, alpha, &flattened_line_coeffs[sampled_offset], &flattened_line_coeffs[sampled_offset + 1], &flattened_line_coeffs[sampled_offset + 2]); 
        }
    }
}

// __device__ void denominator_inverse(point domain_point, column_sample_batch *sample_batches, cm31 *result) {
//     result[0] = {1234450342, 2089936180}; // Result of denominator_inverse(sample_batches, domain.at(0))
// }

__device__ void denominator_inverse(
        column_sample_batch *sample_batches,
        uint32_t sample_size,
        const point domain_point,
        cm31 *flat_denominators) {

    for (unsigned int i = 0; i < sample_size; i++) {
        cm31 prx = sample_batches[i].point.x.a;
        cm31 pry = sample_batches[i].point.y.a;
        cm31 pix = sample_batches[i].point.x.b;
        cm31 piy = sample_batches[i].point.y.b;

        cm31 first_substraction = {sub(prx.a, domain_point.x), prx.b};
        cm31 second_substraction = {sub(pry.a, domain_point.y), pry.b};
        cm31 result = sub(mul(first_substraction, piy),
                          mul(second_substraction, pix));
        flat_denominators[i] = inv(result);
    }
}

__global__ void accumulate_quotients_in_gpu(
        uint32_t half_coset_initial_index,
        uint32_t half_coset_step_size,
        uint32_t domain_size,
        int domain_log_size,
        m31 **columns,
        uint32_t number_of_columns,
        qm31 random_coefficient,
        column_sample_batch *sample_batches,
        uint32_t sample_size,
        uint32_t *result_column_0,
        uint32_t *result_column_1,
        uint32_t *result_column_2,
        uint32_t *result_column_3,
        qm31 *flattened_line_coeffs,
        uint32_t *line_coeffs_sizes,
        qm31 *batch_random_coeffs,
        cm31 *denominator_inverses
) {
    int row = threadIdx.x + blockDim.x * blockIdx.x;
    denominator_inverses = &denominator_inverses[row * sample_size];

    if (row < domain_size) {
        uint32_t domain_index = bit_reverse(row, domain_log_size);
        point domain_point = domain_at_index(half_coset_initial_index, half_coset_step_size, domain_index, domain_size);

        denominator_inverse(
            sample_batches,
            sample_size,
            domain_point,
            denominator_inverses
        );

        int i = 0;

        qm31 row_accumulator = qm31{cm31{0, 0}, cm31{0, 0}};
        int line_coeffs_offset = 0;
        while (i < sample_size) {
            column_sample_batch sample_batch = sample_batches[i];
            qm31 *line_coeffs = &flattened_line_coeffs[line_coeffs_offset * 3];
            qm31 batch_coeff = batch_random_coeffs[i];
            int line_coeffs_size = line_coeffs_sizes[i];

            qm31 numerator = qm31{cm31{0, 0}, cm31{0, 0}};
            for(int j = 0; j < line_coeffs_size; j++) {
                qm31 a = line_coeffs[3 * j + 0];
                qm31 b = line_coeffs[3 * j + 1];
                qm31 c = line_coeffs[3 * j + 2];

                int column_index = sample_batches[i].columns[j];
                qm31 linear_term = add(mul_by_scalar(a, domain_point.y), b);
                qm31 value = mul_by_scalar(c, columns[column_index][row]);

                numerator = add(numerator, sub(value, linear_term));
            }

            row_accumulator = add(mul(row_accumulator, batch_coeff), mul(numerator, denominator_inverses[i]));
            line_coeffs_offset += line_coeffs_size;
            i++;
        }

        result_column_0[row] = row_accumulator.a.a;
        result_column_1[row] = row_accumulator.a.b;
        result_column_2[row] = row_accumulator.b.a;
        result_column_3[row] = row_accumulator.b.b;

    }
}

void accumulate_quotients(
        uint32_t half_coset_initial_index,
        uint32_t half_coset_step_size,
        uint32_t domain_size,
        m31 **columns,
        uint32_t number_of_columns,
        qm31 random_coefficient,
        secure_field_point *sample_points,
        uint32_t *sample_column_indexes,
        uint32_t sample_column_indexes_size,
        qm31 *sample_column_values,
        uint32_t *sample_column_and_values_sizes,
        uint32_t sample_size,
        uint32_t *result_column_0,
        uint32_t *result_column_1,
        uint32_t *result_column_2,
        uint32_t *result_column_3,
        uint32_t flattened_line_coeffs_size
) {
    int domain_log_size = log_2((int)domain_size);

    auto sample_batches = (column_sample_batch *)malloc(sizeof(column_sample_batch) * sample_size);

    column_sample_batch *sample_batches_device;
    hipMalloc((void**)&sample_batches_device, sizeof(column_sample_batch) * sample_size);
    cm31* denominator_inverses;

    hipMalloc((void**)&denominator_inverses, sizeof(cm31) * sample_size * domain_size);

    uint32_t *sample_column_indexes_device;
    hipMalloc((void**)&sample_column_indexes_device, sizeof(uint32_t) * sample_column_indexes_size);
    hipMemcpy(sample_column_indexes_device, sample_column_indexes, sizeof(uint32_t) * sample_column_indexes_size, hipMemcpyHostToDevice);

    qm31 *sample_column_values_device;
    hipMalloc((void**)&sample_column_values_device, sizeof(qm31) * sample_column_indexes_size);
    hipMemcpy(sample_column_values_device, sample_column_values, sizeof(qm31) * sample_column_indexes_size, hipMemcpyHostToDevice);

    column_sample_batches_for(
            sample_points,
            sample_column_indexes_device,
            sample_column_values_device,
            sample_column_and_values_sizes,
            sample_size,
            sample_batches
    );

    hipMemcpy(sample_batches_device, sample_batches, sizeof(column_sample_batch) * sample_size, hipMemcpyHostToDevice);

    qm31 *batch_random_coeffs_device;
    hipMalloc((void**)&batch_random_coeffs_device, sizeof(qm31) * sample_size);

    uint32_t *line_coeffs_sizes_device;
    hipMalloc((void**)&line_coeffs_sizes_device, sizeof(uint32_t) * sample_size);

    qm31 *flattened_line_coeffs_device;
    hipMalloc((void**)&flattened_line_coeffs_device, sizeof(qm31) * flattened_line_coeffs_size);

    // Accumulate Quotient Constants
    int block_dim = sample_size < THREAD_MAX_COUNT ? sample_size : THREAD_MAX_COUNT; 
    int num_blocks = block_dim < THREAD_MAX_COUNT ? 1 : (sample_size + block_dim - 1) / block_dim;
    column_line_and_batch_random_coeffs<<<num_blocks, block_dim>>>(
            sample_batches_device, 
            sample_size, 
            random_coefficient,
            flattened_line_coeffs_device, 
            line_coeffs_sizes_device,
            batch_random_coeffs_device
    );

    // TODO: set to 1024
    block_dim = 512;
    num_blocks = (domain_size + block_dim - 1) / block_dim;
    accumulate_quotients_in_gpu<<<num_blocks, block_dim>>>(
            half_coset_initial_index,
            half_coset_step_size,
            domain_size,
            domain_log_size,
            columns,
            number_of_columns,
            random_coefficient,
            sample_batches_device,
            sample_size,
            result_column_0,
            result_column_1,
            result_column_2,
            result_column_3,
            flattened_line_coeffs_device,
            line_coeffs_sizes_device,
            batch_random_coeffs_device,
            denominator_inverses
    );
    hipDeviceSynchronize();

    free(sample_batches);
    hipFree(sample_batches_device);
    hipFree(denominator_inverses);
    hipFree(sample_column_indexes_device);
    hipFree(sample_column_values_device);
    hipFree(batch_random_coeffs_device);
    hipFree(line_coeffs_sizes_device);
    hipFree(flattened_line_coeffs_device);
}
