#include "hip/hip_runtime.h"
#include "fri/fold_line.cuh"
#include "fri/utils.cuh"

__device__ uint32_t f(const uint32_t *domain,
                      const uint32_t twiddle_offset,
                      const uint32_t i) {
    return domain[i + twiddle_offset];
}

__global__ void fold_applying(const uint32_t *domain,
                              const uint32_t twiddle_offset,
                              const uint32_t n,
                              const qm31 alpha,
                              uint32_t *eval_values_0,
                              uint32_t *eval_values_1,
                              uint32_t *eval_values_2,
                              uint32_t *eval_values_3,
                              uint32_t *folded_values_0,
                              uint32_t *folded_values_1,
                              uint32_t *folded_values_2,
                              uint32_t *folded_values_3) {
    const uint32_t *eval_values[4] = {eval_values_0,
                                      eval_values_1,
                                      eval_values_2,
                                      eval_values_3};

    const uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n / 2) {
        const uint32_t x_inverse = f(domain, twiddle_offset, i);

        const uint32_t index_left = 2 * i;
        const uint32_t index_right = index_left + 1;

        const qm31 f_x = getEvaluation(eval_values, index_left);
        const qm31 f_x_minus = getEvaluation(eval_values, index_right);

        const qm31 f_0 = add(f_x, f_x_minus);
        const qm31 f_1 = mul_by_scalar(sub(f_x, f_x_minus), x_inverse);

        const qm31 f_prime = add(f_0, mul(alpha, f_1));

        folded_values_0[i] = f_prime.a.a;
        folded_values_1[i] = f_prime.a.b;
        folded_values_2[i] = f_prime.b.a;
        folded_values_3[i] = f_prime.b.b;
    }
}

void fold_line(uint32_t *gpu_domain,
               uint32_t twiddle_offset,
               uint32_t n,
               uint32_t **eval_values,
               qm31 alpha,
               uint32_t **folded_values) {
    int block_dim = 1024;
    int num_blocks = (n / 2 + block_dim - 1) / block_dim;
    fold_applying<<<num_blocks, block_dim>>>(
            gpu_domain,
            twiddle_offset,
            n,
            alpha,
            eval_values[0],
            eval_values[1],
            eval_values[2],
            eval_values[3],
            folded_values[0],
            folded_values[1],
            folded_values[2],
            folded_values[3]);
    hipDeviceSynchronize();
}

