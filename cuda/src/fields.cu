#include "hip/hip_runtime.h"
#include "../include/fields.cuh"

__host__ __device__ m31 mul(m31 a, m31 b) {
    uint64_t v = ((uint64_t) a * (uint64_t) b);
    uint64_t w = v + (v >> 31);
    uint64_t u = v + (w >> 31);
    return u & P;
}

__host__ __device__ m31 add(m31 a, m31 b) {
    uint64_t sum = ((uint64_t) a + (uint64_t) b);
    return min(sum, sum - P);
}

__host__ __device__ m31 sub(m31 a, m31 b) {
    return add(a, P - b);
}

__host__ __device__ m31 neg(m31 a) {
    return P - a;
}

__host__ __device__ uint64_t pow_to_power_of_two(int n, m31 t) {
    int i = 0;
    while (i < n) {
        t = mul(t, t);
        i++;
    }
    return t;
}

__host__ __device__ m31 inv(m31 t) {
    uint64_t t0 = mul(pow_to_power_of_two(2, t), t);
    uint64_t t1 = mul(pow_to_power_of_two(1, t0), t0);
    uint64_t t2 = mul(pow_to_power_of_two(3, t1), t0);
    uint64_t t3 = mul(pow_to_power_of_two(1, t2), t0);
    uint64_t t4 = mul(pow_to_power_of_two(8, t3), t3);
    uint64_t t5 = mul(pow_to_power_of_two(8, t4), t3);
    return mul(pow_to_power_of_two(7, t5), t2);
}

/*##### CM31 ##### */

__host__ __device__ cm31 mul(cm31 x, cm31 y) {
    return {sub(mul(x.a, y.a), mul(x.b, y.b)), add(mul(x.a, y.b), mul(x.b, y.a))};
}

__host__ __device__ cm31 add(cm31 x, cm31 y) {
    return {add(x.a, y.a), add(x.b, y.b)};
}

__host__ __device__ cm31 sub(cm31 x, cm31 y) {
    return {sub(x.a, y.a), sub(x.b, y.b)};
}

__host__ __device__ cm31 neg(cm31 x) {
    return {neg(x.a), neg(x.b)};
}

__host__ __device__ cm31 inv(cm31 t) {
    m31 factor = inv(add(mul(t.a, t.a), mul(t.b, t.b)));
    return {mul(t.a, factor), mul(neg(t.b), factor)};
}

__host__ __device__ cm31 mul_by_scalar(cm31 x, m31 scalar) {
    return cm31 { mul(x.a, scalar), mul(x.b, scalar) };
}

/*##### QM31 ##### */

__host__ __device__ qm31 mul(qm31 x, qm31 y) {
    // Karatsuba multiplication
    cm31 v0 = mul(x.a, y.a);
    cm31 v1 = mul(x.b, y.b);
    cm31 v2 = mul(add(x.a, x.b), add(y.a, y.b));
    return {
            add(v0, mul(R, v1)),
            sub(v2, add(v0, v1))
    };
}

__host__ __device__ qm31 add(qm31 x, qm31 y) {
    return {add(x.a, y.a), add(x.b, y.b)};
}

__host__ __device__ qm31 sub(qm31 x, qm31 y) {
    return {sub(x.a, y.a), sub(x.b, y.b)};
}

__host__ __device__ qm31 mul_by_scalar(qm31 x, m31 scalar) {
    return qm31 { mul_by_scalar(x.a, scalar), mul_by_scalar(x.b, scalar) };
}

__host__ __device__ qm31 inv(qm31 t) {
    cm31 b2 = mul(t.b, t.b);
    cm31 ib2 = {neg(b2.b), b2.a};
    cm31 denom = sub(mul(t.a, t.a), add(add(b2, b2), ib2));
    cm31 denom_inverse = inv(denom);
    return {mul(t.a, denom_inverse), neg(mul(t.b, denom_inverse))};
}


const uint32_t MODULUS = (1 << 31) - 1; 

__device__ M31 M31::zero() {
    return M31(); 
}

__device__ M31 M31::one() {
    return M31(1); 
}

__device__  M31 M31::operator+(const M31& rhs) const {
    uint32_t out = f + rhs.f; 
    return M31((uint32_t)min(out, out - MODULUS));
}

__device__  M31 M31::operator-(const M31& rhs) const {
    uint32_t out = f - rhs.f;
    return M31((uint32_t)min(out, out + MODULUS));
}

__device__  M31 M31::operator-() const {
    return M31(MODULUS - f);
}

__device__  M31 M31::operator*(const M31& rhs) const {
    unsigned long long int a_e, b_e, prod_e;
    uint32_t prod_lows, prod_highs;

    a_e = (unsigned long long int) f;
    b_e = (unsigned long long int) rhs.f;

    prod_e = a_e * b_e;
    prod_lows = (unsigned long long int) prod_e & 0x7FFFFFFF;
    prod_highs = (unsigned long long int) prod_e >> 31;

    uint32_t out = prod_lows + prod_highs; 
    return M31((uint32_t) min(out, out - MODULUS));
    // uint64_t v = ((uint64_t) f * (uint64_t) rhs.f);
    // uint64_t w = v + (v >> 31);
    // uint64_t u = v + (w >> 31);
    // return M31(u & P);
}

__device__ CM31::CM31() : a(0), b(0) {}
__device__ CM31::CM31(M31 a, M31 b) : a(a), b(b) {}
__device__ CM31::CM31(uint32_t a, uint32_t b) : a(M31(a)), b(M31(b)) {} 

__device__ CM31 CM31::zero() {
    return CM31(); 
}

__device__ CM31 CM31::one() {
    return CM31(M31::one(), M31::zero()); 
}
__device__ CM31 CM31::operator*(const CM31& rhs) const {
    return CM31(
        a * rhs.a - b * rhs.b,
        a * rhs.b + b * rhs.a 
    );
}

__device__ CM31 CM31::operator+(const CM31& rhs) const {
    return CM31(a + rhs.a, b + rhs.b);
}

__device__ CM31 CM31::operator-(const CM31& rhs) const {
    return CM31(a - rhs.a, b - rhs.b);
}

__device__ CM31 CM31::operator-() const {
    return CM31(-a, -b);
}

__device__ CM31 CM31::operator+(const M31& rhs) const {
    return CM31(a + rhs, b); 
}

__device__ QM31::QM31() : a(CM31()), b(CM31()) {}
__device__ QM31::QM31(CM31 a, CM31 b) : a(a), b(b) {}

__device__ QM31 QM31::zero() {
    return QM31();
}

__device__ QM31 QM31::one() {
    return QM31(CM31::one(), CM31::zero()); 
}

__device__ QM31 QM31::operator+(const M31& rhs) const {
    return QM31(a + rhs, b); 
}

__device__ QM31 QM31::operator*(const QM31& rhs) const {
    return QM31(
        a * rhs.a + CM31(M31(2), M31(1)) * b * rhs.b,
        a * rhs.b + b * rhs.a
    );
}

 __device__ QM31 QM31::operator-() const {
     return QM31(-a, -b);
 }

 __device__ QM31 QM31::operator+(const QM31& rhs) const {
     return QM31(a + rhs.a, b + rhs.b);
 }

 __device__ QM31 QM31::operator-(const QM31& rhs) const {
     return QM31(a - rhs.a, b - rhs.b);
 }

 __device__ QM31 square(const QM31& self) {
    return self * self;
 }

// u128?
 __device__ QM31 pow(const QM31& self, uint64_t exp) {
    QM31 res = QM31::one();
    QM31 base = self;
    while (exp > 0) {
            if (exp & 1) {
                res = res * base;
            }
            base = square(base);
            exp >>= 1;
        }
    return res; 
 }

// Acc row quotients helper
 __device__ QM31 sub_from_m31(const M31& lhs, const QM31& rhs) {
     return -rhs + lhs;
 }
