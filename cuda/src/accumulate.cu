#include "hip/hip_runtime.h"
#include "../include/accumulate.cuh"

__global__ void
accumulate_aux(uint32_t *left_column_0, uint32_t *left_column_1, uint32_t *left_column_2, uint32_t *left_column_3,
               uint32_t *right_column_0, uint32_t *right_column_1, uint32_t *right_column_2,
               uint32_t *right_column_3);

void accumulate(uint32_t size, uint32_t *left_column_0, uint32_t *left_column_1, uint32_t *left_column_2,
                uint32_t *left_column_3, uint32_t *right_column_0, uint32_t *right_column_1, uint32_t *right_column_2,
                uint32_t *right_column_3) {
    int block_dim = 1024;
    int num_blocks = (size + block_dim - 1) / block_dim;
    accumulate_aux<<<num_blocks, min(size, block_dim)>>>(
            left_column_0, left_column_1, left_column_2, left_column_3,
            right_column_0, right_column_1, right_column_2, right_column_3);
    hipDeviceSynchronize();
}

__global__ void
accumulate_aux(uint32_t *left_column_0, uint32_t *left_column_1, uint32_t *left_column_2, uint32_t *left_column_3,
               uint32_t *right_column_0, uint32_t *right_column_1, uint32_t *right_column_2,
               uint32_t *right_column_3) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    left_column_0[i] += right_column_0[i];
    left_column_1[i] += right_column_1[i];
    left_column_2[i] += right_column_2[i];
    left_column_3[i] += right_column_3[i];
}
