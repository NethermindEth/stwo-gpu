#include "hip/hip_runtime.h"
#include "poly/rfft.cuh"
#include "poly/utils.cuh"
#include "utils.cuh"

__global__ void rfft_circle_part(m31 *values, m31 *inverse_twiddles_tree, int values_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;


    if (idx < (values_size >> 1)) {
        m31 val0 = values[2 * idx];
        m31 val1 = values[2 * idx + 1];
        m31 twiddle = get_twiddle(inverse_twiddles_tree, idx);

        m31 temp = mul(val1, twiddle);

        values[2 * idx] = add(val0, temp);
        values[2 * idx + 1] = sub(val0, temp);
    }
}

__global__ void rfft_line_part(m31 *values, m31 *inverse_twiddles_tree, int values_size, int inverse_twiddles_size,
                               int layer_domain_offset, int layer) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < (values_size >> 1)) {
        int number_polynomials = 1 << layer;
        int h = idx / number_polynomials;
        int l = idx % number_polynomials;
        int idx0 = (h << (layer + 1)) + l;
        int idx1 = idx0 + number_polynomials;

        m31 val0 = values[idx0];
        m31 val1 = values[idx1];
        m31 twiddle = inverse_twiddles_tree[layer_domain_offset + h];

        m31 temp = mul(val1, twiddle);

        values[idx0] = add(val0, temp);
        values[idx1] = sub(val0, temp);
    }
}



void evaluate(int eval_domain_size, m31 *values, m31 *twiddles_tree, int twiddles_size, int values_size) {
    twiddles_tree = &twiddles_tree[twiddles_size - eval_domain_size];
    int block_dim = 256;
    int num_blocks = ((values_size >> 1) + block_dim - 1) / block_dim;

    int log_values_size = log_2(values_size);
    int layer_domain_size = 1;
    int layer_domain_offset = (values_size >> 1) - 2;
    int i = log_values_size - 1;
    while (i > 0) {
        rfft_line_part<<<num_blocks, block_dim>>>(values, twiddles_tree, values_size, layer_domain_size,
                                                  layer_domain_offset, i);
        layer_domain_size <<= 1;
        layer_domain_offset -= layer_domain_size;
        i -= 1;
    }

    rfft_circle_part<<<num_blocks, block_dim>>>(values, twiddles_tree, values_size);
    hipDeviceSynchronize();
}
