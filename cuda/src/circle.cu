#include "hip/hip_runtime.h"
#include "../include/circle.cuh"

__global__ void sort_values_kernel(m31 *from, m31 *dst, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        if(idx < (size >> 1)) {
            dst[idx] = from[idx << 1];
        } else {
            int tmp = idx - (size >> 1);
            dst[idx] = from[size - (tmp << 1) - 1];
        }
    }
}

m31* sort_values(m31 *from, int size) {
    int block_dim = 256;
    int num_blocks = (size + block_dim - 1) / block_dim;
    m31 *dst;
    hipMalloc((void**)&dst, sizeof(m31) * size);
    sort_values_kernel<<<num_blocks, block_dim>>>(from, dst, size);
    hipDeviceSynchronize();
    return dst;
}
