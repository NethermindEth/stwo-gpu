#include "../include/utils.cuh"

void copy_uint32_t_vec_from_device_to_host(uint32_t *device_ptr, uint32_t *host_ptr, int size) {
    hipMemcpy(host_ptr, device_ptr, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
}

uint32_t* copy_uint32_t_vec_from_host_to_device(uint32_t *host_ptr, int size) {
    uint32_t* device_ptr;
    hipMalloc((void**)&device_ptr, sizeof(uint32_t) * size);
    hipMemcpy(device_ptr, host_ptr, sizeof(uint32_t) * size, hipMemcpyHostToDevice);
    return device_ptr;
}

void copy_uint32_t_vec_from_device_to_device(uint32_t *from, uint32_t *dst, int size) {
    hipMemcpy(dst, from, sizeof(uint32_t) * size, hipMemcpyDeviceToDevice);
}

uint32_t* cuda_malloc_uint32_t(int size) {
    uint32_t* device_ptr;
    hipMalloc((void**)&device_ptr, sizeof(uint32_t) * size);
    return device_ptr;
}

uint32_t* cuda_alloc_zeroes_uint32_t(int size) {
    uint32_t* device_ptr = cuda_malloc_uint32_t(size);
    hipMemset(device_ptr, 0x00, sizeof(uint32_t) * size);
    return device_ptr;
}

void free_uint32_t_vec(uint32_t *device_ptr) {
    hipFree(device_ptr);
}
