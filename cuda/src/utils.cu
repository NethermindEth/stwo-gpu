#include "hip/hip_runtime.h"
#include "../include/utils.cuh"

__global__ void initialize_memory(int* device_array, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < size) {
        device_array[tid] = tid;
    }
}

extern "C"
void copy_m31_vec_from_device_to_host(uint32_t *device_ptr, uint32_t *host_ptr, int size) {
    hipMemcpy(host_ptr, device_ptr, sizeof(int) * size, hipMemcpyDeviceToHost);
}

extern "C"
void free_uint32_t_vec(uint32_t *device_ptr) {
    hipFree(device_ptr);
}

extern "C"
int* generate_array(int size) {
    int *device_array;
    hipMalloc((void**)&device_array, size * sizeof(int));
    initialize_memory<<<256, 512>>>(device_array, size);
    hipDeviceSynchronize();
    return device_array;
}

extern "C"
int sum(int *device_array, int size) {
    int* host_array = (int*)malloc(size * sizeof(int));
    hipMemcpy(host_array, device_array, sizeof(int) * size, hipMemcpyDeviceToHost);
    return host_array[size - 1];
}
