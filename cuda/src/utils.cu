#include "../include/utils.cuh"

void copy_uint32_t_vec_from_device_to_host(uint32_t *device_ptr, uint32_t *host_ptr, int size) {
    hipMemcpy(host_ptr, device_ptr, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
}

uint32_t* copy_uint32_t_vec_from_host_to_device(uint32_t *host_ptr, int size) {
    uint32_t* device_ptr;
    hipMalloc((void**)&device_ptr, sizeof(uint32_t) * size);
    hipMemcpy(device_ptr, host_ptr, sizeof(uint32_t) * size, hipMemcpyHostToDevice);
    return device_ptr;
}

uint32_t* cuda_malloc_uint32_t(int size) {
    uint32_t* device_ptr;
    hipMalloc((void**)&device_ptr, sizeof(uint32_t) * size);
    return device_ptr;
}

void free_uint32_t_vec(uint32_t *device_ptr) {
    hipFree(device_ptr);
}
