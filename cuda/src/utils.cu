#include "hip/hip_runtime.h"
#include "../include/utils.cuh"

__global__ void initialize_memory(int* device_array, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < size) {
        device_array[tid] = tid;
    }
}

extern "C"
void copy_m31_vec_from_device_to_host(uint32_t *device_ptr, uint32_t *host_ptr, int size) {
    hipMemcpy(host_ptr, device_ptr, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
}

extern "C"
uint32_t* copy_m31_vec_from_host_to_device(uint32_t *host_ptr, int size) {
    uint32_t* device_ptr;
    hipMalloc((void**)&device_ptr, sizeof(uint32_t) * size);
    hipMemcpy(device_ptr, host_ptr, sizeof(uint32_t) * size, hipMemcpyHostToDevice);
    return device_ptr;
}

extern "C"
void free_uint32_t_vec(uint32_t *device_ptr) {
    hipFree(device_ptr);
}

extern "C"
int* generate_array(int size) {
    int *device_array;
    hipMalloc((void**)&device_array, size * sizeof(int));
    initialize_memory<<<256, 512>>>(device_array, size);
    hipDeviceSynchronize();
    return device_array;
}

extern "C"
int last(int *device_array, int size) {
    int* host_array = (int*)malloc(size * sizeof(int));
    hipMemcpy(host_array, device_array, sizeof(int) * size, hipMemcpyDeviceToHost);
    return host_array[size - 1];
}
