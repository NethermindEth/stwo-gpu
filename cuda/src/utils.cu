#include "../include/utils.cuh"

extern "C"
void copy_uint32_t_vec_from_device_to_host(uint32_t *device_ptr, uint32_t *host_ptr, int size) {
    hipMemcpy(host_ptr, device_ptr, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
}

extern "C"
uint32_t* copy_uint32_t_vec_from_host_to_device(uint32_t *host_ptr, int size) {
    uint32_t* device_ptr;
    hipMalloc((void**)&device_ptr, sizeof(uint32_t) * size);
    hipMemcpy(device_ptr, host_ptr, sizeof(uint32_t) * size, hipMemcpyHostToDevice);
    return device_ptr;
}

extern "C"
void free_uint32_t_vec(uint32_t *device_ptr) {
    hipFree(device_ptr);
}
