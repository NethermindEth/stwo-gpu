#include "hip/hip_runtime.h"
#include "../include/fri.cuh"
#include "../include/utils.cuh"

__device__ void sum_block_list(uint32_t *results, const uint32_t block_thread_index, const uint32_t half_list_size,
                               const uint32_t *list_to_sum_in_block, uint32_t &thread_result) {
    uint32_t list_to_sum_in_block_half_size = min(half_list_size, blockDim.x) >> 1;
    while (block_thread_index < list_to_sum_in_block_half_size) {
        thread_result = add(
                thread_result, list_to_sum_in_block[block_thread_index + list_to_sum_in_block_half_size]);

        __syncthreads();

        list_to_sum_in_block_half_size >>= 1;
    }

    const bool is_first_thread_in_block = block_thread_index == 0;
    if (is_first_thread_in_block) {
        results[blockIdx.x] = thread_result;
    }
}

__global__ void sum_reduce(uint32_t *list, uint32_t *temp_list, uint32_t *results, const uint32_t list_size) {
    const uint32_t block_thread_index = threadIdx.x;
    const uint32_t first_thread_in_block_index = blockIdx.x * blockDim.x;
    const uint32_t grid_thread_index = first_thread_in_block_index + block_thread_index;
    const uint32_t half_list_size = list_size >> 1;

    if (grid_thread_index < half_list_size) {
        uint32_t *list_to_sum_in_block = &temp_list[first_thread_in_block_index];
        uint32_t &thread_result = list_to_sum_in_block[block_thread_index];

        thread_result = sub(
                list[grid_thread_index],
                list[grid_thread_index + half_list_size]);

        __syncthreads();

        sum_block_list(results, block_thread_index, half_list_size, list_to_sum_in_block, thread_result);
    }
}

extern "C"
uint32_t sum(uint32_t *list, const uint32_t list_size) {
    int block_dim = 1024;
    int num_blocks = (list_size / 2 + block_dim - 1) / block_dim;

    uint32_t* temp_list = cuda_malloc_uint32_t(list_size);
    uint32_t* partial_results = cuda_alloc_zeroes_uint32_t(num_blocks);
    sum_reduce<<<num_blocks, min(list_size, block_dim)>>>(list, temp_list, partial_results, list_size);

    uint32_t* results = (uint32_t*) malloc(sizeof(uint32_t)*num_blocks);
    copy_uint32_t_vec_from_device_to_host(partial_results, results, num_blocks);
    free_uint32_t_vec(temp_list);
    free_uint32_t_vec(partial_results);
    uint32_t result = 0;
    for(uint32_t i=0; i < num_blocks; i++) {
        result = add(result, results[i]);
    }
    return result;
}

__global__ void compute_g_values_aux(uint32_t *f_values, uint32_t *results, int size, uint32_t lambda) {
    // Computes one coordinate of the QM31 g_values for the decomposition f = g + lambda * v_n at the first step of FRI.
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        if (idx < (size >> 1)) {
            results[idx] = sub(f_values[idx], lambda);
        }
        if (idx >= (size >> 1)) {
            results[idx] = add(f_values[idx], lambda);
        }
    }
}


uint32_t* compute_g_values(uint32_t *f_values, uint32_t size, uint32_t lambda) {
    int block_dim = 1024;
    int num_blocks = (size + block_dim - 1) / block_dim;
    uint32_t* results = cuda_alloc_zeroes_uint32_t(size);
    compute_g_values_aux<<<num_blocks, min(size, block_dim)>>>(f_values, results, size, lambda);

    return results;
}