
#include <hip/hip_runtime.h>
typedef unsigned int uint32_t;

extern "C"
__global__ void sort_values(uint32_t *from, uint32_t *dst, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        if(idx < (size >> 1)) {
            dst[idx] = from[idx << 1];
        } else {
            int tmp = idx - (size >> 1);
            dst[idx] = from[size - (tmp << 1) - 1];
        }
    }
}