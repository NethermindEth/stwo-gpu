#include "hip/hip_runtime.h"
#include "m31.h"

__device__ __constant__ int MODULUS = (1 << 31) - 1; 

// TODO: Check if using Shared memory per block over device for optimizations
extern "C" __global__  void mul_m31(unsigned int lhs, unsigned int rhs, unsigned int *out) {
    unsigned long long int a_e;
    unsigned long long int b_e;
    unsigned long long int prod_e;
    unsigned int prod_lows;
    unsigned int prod_highs;

    a_e = static_cast<unsigned long long int>(lhs);
    b_e = static_cast<unsigned long long int>(rhs);

    prod_e = a_e * b_e;
    
    // TODO:: look at optimizing through union (check performance)
    prod_lows = static_cast<unsigned int>(prod_e & 0x7FFFFFFF);

    prod_highs = static_cast<unsigned int>(prod_e >> 31);

    // add 
    *out = prod_lows + prod_highs; 
    *out = min(*out, *out - MODULUS);
}

extern "C" __global__  void add_m31(unsigned int lhs,  unsigned int rhs, unsigned int *out) {
    *out = lhs + rhs; 
    
    *out = min(*out, *out - MODULUS);
}

extern "C" __global__ void reduce_m31(unsigned int *f) {
    *f = min(*f, *f - MODULUS);
}

extern "C" __global__  void sub_m31(unsigned int lhs, unsigned int rhs, unsigned int *out) {
    *out = lhs - rhs; 
    *out = min(*out, *out + MODULUS);
}

extern "C" __global__  void neg_m31(unsigned int *f) {
    *f = MODULUS - *f;
}

extern "C" __global__ void mul(unsigned int *a, unsigned int *b, unsigned int *out, int size) {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) {
        mul_m31(a[tid], b[tid], &out[tid]);
    }
}

extern "C" __global__ void add(unsigned int *a, unsigned int *b, unsigned int *out, int size) {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        add_m31(a[tid], b[tid], &out[tid]);
    }
}

extern "C" __global__ void reduce(unsigned int *out, int size) {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) {
        reduce_m31(&out[tid]);
    }
}

extern "C" __global__ void sub(unsigned int *a, unsigned int *b, unsigned int *out, int size) {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) {
        sub_m31(a[tid], b[tid], &out[tid]);
    }
}

extern "C" __global__ void neg(unsigned int *a, int size) {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) {
        neg_m31(&a[tid]);
    }
}