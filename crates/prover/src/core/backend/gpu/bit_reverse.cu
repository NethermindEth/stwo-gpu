
#include <hip/hip_runtime.h>
__device__ unsigned int bit_reverse(unsigned int n, int bits) {
    unsigned int reversed_n = __brev(n);
    return reversed_n >> (32 - bits);
}

extern "C"
__global__ void kernel(int *d_array, int size, int bits) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

        unsigned int rev_idx = bit_reverse(idx, bits);

        if (rev_idx > idx && idx < size) {
            int temp = d_array[idx];
            d_array[idx] = d_array[rev_idx];
            d_array[rev_idx] = temp;
        }
}