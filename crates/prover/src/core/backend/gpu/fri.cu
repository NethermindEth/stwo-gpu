#include "hip/hip_runtime.h"
typedef unsigned int uint32_t;
typedef unsigned long long uint64_t;

typedef struct {
    uint32_t a;
    uint32_t b;
} cm31;

typedef struct {
    cm31 a;
    cm31 b;
} qm31;

const uint32_t P = 2147483647;
const cm31 R = {2, 1};

/*##### M31 ##### */

__device__ uint32_t m31_mul(uint32_t a, uint32_t b) {
    // TODO: use mul from m31.cu
    uint64_t v = ((uint64_t) a * (uint64_t) b);
    uint64_t w = v + (v >> 31);
    uint64_t u = v + (w >> 31);
    return u & P;
}

__device__ uint32_t m31_add(uint32_t a, uint32_t b) {
    // TODO: use add from m31.cu
    return ((uint64_t) a + (uint64_t) b) % P;
}

__device__ uint32_t m31_sub(uint32_t a, uint32_t b) {
    // TODO: use sub from m31.cu
    return ((uint64_t) a + (uint64_t) (P - b)) % P;
}

__device__ uint32_t m31_neg(uint32_t a) {
    // TODO: use sub from m31.cu
    return P - a;
}


/*##### CM1 ##### */

__device__ cm31 cm31_mul(cm31 x, cm31 y) {
    return {m31_sub(m31_mul(x.a, y.a), m31_mul(x.b, y.b)), m31_add(m31_mul(x.a, y.b), m31_mul(x.b, y.a))};
}

__device__ cm31 cm31_add(cm31 x, cm31 y) {
    return {m31_add(x.a, y.a), m31_add(x.b, y.b)};
}

__device__ cm31 cm31_sub(cm31 x, cm31 y) {
    return {m31_sub(x.a, y.a), m31_sub(x.b, y.b)};
}

__device__ cm31 cm31_scalar_mul(cm31 x, uint32_t y) {
    return {
            m31_mul(x.a, y),
            m31_mul(x.b, y),
    };
}

/*##### Q31 ##### */

__device__ qm31 qm31_mul(qm31 x, qm31 y) {
    return {
            cm31_add(
                    cm31_mul(x.a, y.a),
                    cm31_mul(R, cm31_mul(x.b, y.b))
            ),
            cm31_add(
                    cm31_mul(x.a, y.b),
                    cm31_mul(x.b, y.a)
            )
    };
}

__device__ qm31 qm31_add(qm31 x, qm31 y) {
    return {cm31_add(x.a, y.a), cm31_add(x.b, y.b)};
}

__device__ qm31 qm31_sub(qm31 x, qm31 y) {
    return {cm31_sub(x.a, y.a), cm31_sub(x.b, y.b)};
}

__device__ qm31 qm31_scalar_mul(qm31 x, uint32_t y) {
    return {
            cm31_scalar_mul(x.a, y),
            cm31_scalar_mul(x.b, y),
    };
}

/* ##### Repeated from circle.cu (to avoid including .cuh header files) ##### */

__device__ const int get_twiddle(const uint32_t *twiddles, const int index) {
    int k = index >> 2;
    if (index % 4 == 0) {
        return twiddles[2 * k + 1];
    } else if (index % 4 == 1) {
        return m31_neg(twiddles[2 * k + 1]);
    } else if (index % 4 == 2) {
        return m31_neg(twiddles[2 * k]);
    } else {
        return twiddles[2 * k];
    }
}

////////////

extern "C"
__device__ void sum_reduce(uint32_t *list, uint32_t *temp_list, uint32_t *results, const uint32_t list_size,
                           uint32_t (*first_reduce_operation)(uint32_t a, uint32_t b)) {
    const uint32_t block_thread_index = threadIdx.x;
    const uint32_t first_thread_in_block_index = blockIdx.x * blockDim.x;
    const uint32_t grid_thread_index = first_thread_in_block_index + block_thread_index;
    const uint32_t half_list_size = list_size >> 1;

    if (grid_thread_index < half_list_size) {
        uint32_t *list_to_sum_in_block = &temp_list[first_thread_in_block_index];
        uint32_t &thread_result = list_to_sum_in_block[block_thread_index];

        thread_result = first_reduce_operation(
                list[grid_thread_index],
                list[grid_thread_index + half_list_size]);

        __syncthreads();

        uint32_t list_to_sum_in_block_half_size = min(half_list_size, blockDim.x) >> 1;
        while (block_thread_index < list_to_sum_in_block_half_size) {
            thread_result = m31_add(
                    thread_result, list_to_sum_in_block[block_thread_index + list_to_sum_in_block_half_size]);

            __syncthreads();

            list_to_sum_in_block_half_size >>= 1;
        }

        const bool is_first_thread_in_block = block_thread_index == 0;
        if (is_first_thread_in_block) {
            results[blockIdx.x] = thread_result;
        }
    }
}

extern "C"
__global__ void sum(uint32_t *list, uint32_t *temp_list, uint32_t *results, const uint32_t list_size) {
    sum_reduce(list, temp_list, results, list_size, m31_sub);
}

extern "C"
__global__ void pairwise_sum(uint32_t *list, uint32_t *temp_list, uint32_t *results, const uint32_t list_size) {
    sum_reduce(list, temp_list, results, list_size, m31_add);
}

extern "C"
__global__ void compute_g_values(uint32_t *f_values, uint32_t *results, uint32_t lambda, int size) {
    // Computes one coordinate of the QM31 g_values for the decomposition f = g + lambda * v_n at the first step of FRI.
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        if (idx < (size >> 1)) {
            results[idx] = m31_sub(f_values[idx], lambda);
        }
        if (idx >= (size >> 1)) {
            results[idx] = m31_add(f_values[idx], lambda);
        }
    }
}

extern "C"
__device__ const uint32_t f(const uint32_t *domain,
                            const uint32_t twiddle_offset,
                            const uint32_t i) {
    return domain[i + twiddle_offset];
}

extern "C"
__device__ const uint32_t g(const uint32_t *domain,
                            const uint32_t _twiddle_offset,
                            const uint32_t i) {
    return get_twiddle(domain, i);
}

extern "C"
__device__ const qm31 getEvaluation(const uint32_t *const *eval_values, const uint32_t index) {
    return {{eval_values[0][index],
                    eval_values[1][index]},
            {eval_values[2][index],
                    eval_values[3][index]}};
}

extern "C"
__device__ void
fold_applying(const uint32_t (*h)(const uint32_t *, const uint32_t, const uint32_t),
              const uint32_t *domain, const uint32_t twiddle_offset,
              const uint32_t n,
              const qm31 &alpha,
              uint32_t **folded_values,
              const uint32_t **eval_values) {
    const uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n / 2) {
        const uint32_t x_inverse = h(domain, twiddle_offset, i);

        const uint32_t index_left = 2 * i;
        const uint32_t index_right = index_left + 1;

        const qm31 f_x = getEvaluation(eval_values, index_left);
        const qm31 f_x_minus = getEvaluation(eval_values, index_right);

        const qm31 f_0 = qm31_add(f_x, f_x_minus);
        const qm31 f_1 = qm31_scalar_mul(qm31_sub(f_x, f_x_minus), x_inverse);

        const qm31 f_prime = qm31_add(f_0, qm31_mul(alpha, f_1));

        folded_values[0][i] = f_prime.a.a;
        folded_values[1][i] = f_prime.a.b;
        folded_values[2][i] = f_prime.b.a;
        folded_values[3][i] = f_prime.b.b;
    }
}

extern "C"
__global__ void fold_line(
        const uint32_t *domain,
        const uint32_t twiddle_offset,
        const uint32_t n,
        const uint32_t *eval_values_0,
        const uint32_t *eval_values_1,
        const uint32_t *eval_values_2,
        const uint32_t *eval_values_3,
        const qm31 alpha,
        uint32_t *folded_values_0,
        uint32_t *folded_values_1,
        uint32_t *folded_values_2,
        uint32_t *folded_values_3
) {
    const uint32_t *eval_values[4] = {eval_values_0, eval_values_1, eval_values_2, eval_values_3};
    uint32_t *folded_values[4] = {folded_values_0, folded_values_1, folded_values_2, folded_values_3};
    fold_applying(f, domain, twiddle_offset, n, alpha, folded_values, eval_values);
}

extern "C"
__global__ void fold_circle_into_line(
        const uint32_t *domain,
        const uint32_t twiddle_offset,
        const uint32_t n,
        const uint32_t *eval_values_0,
        const uint32_t *eval_values_1,
        const uint32_t *eval_values_2,
        const uint32_t *eval_values_3,
        const qm31 alpha,
        uint32_t *folded_values_0,
        uint32_t *folded_values_1,
        uint32_t *folded_values_2,
        uint32_t *folded_values_3
) {
    const uint32_t *eval_values[4] = {eval_values_0, eval_values_1, eval_values_2, eval_values_3};
    uint32_t *folded_values[4] = {folded_values_0, folded_values_1, folded_values_2, folded_values_3};
    fold_applying(g, domain, twiddle_offset, n, alpha, folded_values, eval_values);
}
