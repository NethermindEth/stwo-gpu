#include "hip/hip_runtime.h"
typedef unsigned int uint32_t;
typedef unsigned long long uint64_t;

typedef struct {
    uint32_t a;
    uint32_t b;
} cm31;

typedef struct {
    cm31 a;
    cm31 b;
} qm31;

const uint32_t P = 2147483647;
const cm31 R = {2, 1};

/*##### M31 ##### */

__device__ uint32_t m31_mul(uint32_t a, uint32_t b) {
    // TODO: use mul from m31.cu
    uint64_t v = ((uint64_t) a * (uint64_t) b);
    uint64_t w = v + (v >> 31);
    uint64_t u = v + (w >> 31);
    return u & P;}

__device__ uint32_t m31_add(uint32_t a, uint32_t b) {
    // TODO: use add from m31.cu
    return ((uint64_t) a + (uint64_t) b) % P;
}

__device__ uint32_t m31_sub(uint32_t a, uint32_t b) {
    // TODO: use sub from m31.cu
    return ((uint64_t) a + (uint64_t) (P - b)) % P;
}

__device__ uint32_t m31_neg(uint32_t a) {
    // TODO: use sub from m31.cu
    return P - a;
}


/*##### CM1 ##### */

__device__ cm31 cm31_mul(cm31 x, cm31 y) {
    return {m31_sub(m31_mul(x.a, y.a), m31_mul(x.b, y.b)), m31_add(m31_mul(x.a, y.b), m31_mul(x.b, y.a))};
}

__device__ cm31 cm31_add(cm31 x, cm31 y) {
    return {m31_add(x.a, y.a), m31_add(x.b, y.b)};
}

__device__ cm31 cm31_sub(cm31 x, cm31 y) {
    return {m31_sub(x.a, y.a), m31_sub(x.b, y.b)};
}

/*##### Q31 ##### */

__device__ qm31 qm31_mul(qm31 x, qm31 y) {
    return {
        cm31_add(
            cm31_mul(x.a, y.a),
            cm31_mul(R, cm31_mul(x.b, y.b))
        ),
        cm31_add(
            cm31_mul(x.a, y.b),
            cm31_mul(x.b, y.a)
        )
    };
}

__device__ qm31 qm31_add(qm31 x, qm31 y) {
    return {cm31_add(x.a, y.a), cm31_add(x.b, y.b)};
}

__device__ qm31 qm31_sub(qm31 x, qm31 y) {
    return {cm31_sub(x.a, y.a), cm31_sub(x.b, y.b)};
}

////////////

extern "C"
__device__ void sum_reduce(uint32_t *list, uint32_t* temp_list, uint32_t *results, const uint32_t list_size, uint32_t (*first_reduce_operation)(uint32_t a, uint32_t b)) {
    const uint32_t block_thread_index = threadIdx.x;
    const uint32_t first_thread_in_block_index = blockIdx.x * blockDim.x;
    const uint32_t grid_thread_index = first_thread_in_block_index + block_thread_index;
    const uint32_t half_list_size = list_size >> 1;

    if (grid_thread_index < half_list_size) {
        uint32_t *list_to_sum_in_block = &temp_list[first_thread_in_block_index];
        uint32_t &thread_result = list_to_sum_in_block[block_thread_index];

        thread_result = first_reduce_operation(
            list[grid_thread_index],
            list[grid_thread_index + half_list_size]);

        __syncthreads();

        uint32_t list_to_sum_in_block_half_size = min(half_list_size, blockDim.x) >> 1;
        while(block_thread_index < list_to_sum_in_block_half_size) {
            thread_result = m31_add(
                thread_result, list_to_sum_in_block[block_thread_index + list_to_sum_in_block_half_size]);

            __syncthreads();

            list_to_sum_in_block_half_size >>= 1;
        }

        const bool is_first_thread_in_block = block_thread_index == 0;
        if (is_first_thread_in_block) {
            results[blockIdx.x] = thread_result;
        }
    }
}

extern "C"
__global__ void sum(uint32_t *list, uint32_t* temp_list, uint32_t *results, const uint32_t list_size) {
    sum_reduce(list, temp_list, results, list_size, m31_sub);
}

extern "C"
__global__ void pairwise_sum(uint32_t *list, uint32_t* temp_list, uint32_t *results, const uint32_t list_size) {
    sum_reduce(list, temp_list, results, list_size, m31_add);
}

extern "C"
__global__ void compute_g_values(uint32_t *f_values, uint32_t *results, uint32_t lambda, int size) {
    // Computes one coordinate of the QM31 g_values for the decomposition f = g + lambda * v_n at the first step of FRI.
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < size) {
        if(idx < (size >> 1)) {
            results[idx] = m31_sub(f_values[idx], lambda);
        }
        if(idx >= (size >> 1)) {
            results[idx] = m31_add(f_values[idx], lambda);
        }
    }
}

extern "C"
__global__ void fold_line(
    uint32_t *domain,
    uint32_t twiddle_offset,
    uint32_t n,
    uint32_t *eval_values_0,
    uint32_t *eval_values_1,
    uint32_t *eval_values_2,
    uint32_t *eval_values_3,
    qm31 alpha,
    uint32_t *folded_values_0,
    uint32_t *folded_values_1,
    uint32_t *folded_values_2,
    uint32_t *folded_values_3
) {
    if (blockIdx.x == 0) {
        // TODO: must support list with length bigger than 2^10
        uint32_t i = threadIdx.x;
        if (i < n / 2) {
            uint32_t index_left = 2*i;
            uint32_t index_right = index_left+1;
            
            qm31 f_x = {{eval_values_0[index_left],
                         eval_values_1[index_left]},
                        {eval_values_2[index_left],
                         eval_values_3[index_left]}};
            qm31 f_x_minus = {{eval_values_0[index_right],
                               eval_values_1[index_right]},
                              {eval_values_2[index_right],
                               eval_values_3[index_right]}};
            uint32_t x_inverse = domain[i + twiddle_offset];

            qm31 f_0 = qm31_add(f_x, f_x_minus);
            qm31 f_1_dot_x = qm31_sub(f_x, f_x_minus);
            qm31 f_1 = {
                m31_mul(f_1_dot_x.a.a, x_inverse),
                m31_mul(f_1_dot_x.a.b, x_inverse),
                m31_mul(f_1_dot_x.b.a, x_inverse),
                m31_mul(f_1_dot_x.b.b, x_inverse),
            };

            qm31 f_prime = qm31_add(f_0, qm31_mul(alpha, f_1));

            folded_values_0[i] = f_prime.a.a;
            folded_values_1[i] = f_prime.a.b;
            folded_values_2[i] = f_prime.b.a;
            folded_values_3[i] = f_prime.b.b;
        }
    }
}
